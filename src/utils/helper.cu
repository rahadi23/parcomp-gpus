#include <stdio.h>
#include <stdlib.h>

#include "helper.cuh"

void cudaCheck(hipError_t error_code, const char *file, int line)
{
  if (error_code != hipSuccess)
  {
    fprintf(stderr, "CUDA Error %d: '%s'. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
    fflush(stderr);
    exit(error_code);
  }
}
