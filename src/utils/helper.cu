#include "helper.cuh"

#define CUDACHECK(err)                    \
  do                                      \
  {                                       \
    cudaCheck((err), __FILE__, __LINE__); \
  } while (false)

inline void cudaCheck(hipError_t error_code, const char *file, int line)
{
  if (error_code != hipSuccess)
  {
    fprintf(stderr, "CUDA Error %d: '%s'. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
    fflush(stderr);
    exit(error_code);
  }
}
