
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define CUDACHECK(err)                    \
  do                                      \
  {                                       \
    cudaCheck((err), __FILE__, __LINE__); \
  } while (false)

inline void cudaCheck(hipError_t error_code, const char *file, int line)
{
  if (error_code != hipSuccess)
  {
    fprintf(stderr, "CUDA Error %d: '%s'. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
    fflush(stderr);
    exit(error_code);
  }
}

void parseArgs(int argc, char *argv[], int *dimSize, int *blockSize)
{
  char *cp;
  long lDimSize, lBlockSize;

  // Check for the right number of arguments
  if (argc != 3)
  {
    fprintf(stderr, "[ERROR] Must be run with exactly 2 argument, found %d!\nUsage: %s <N>\n", argc - 1, argv[0]);
    exit(1);
  }

  cp = argv[1];

  if (*cp == 0)
  {
    fprintf(stderr, "[ERROR] Argument '%s' is an empty string\n", argv[1]);
    exit(1);
  }

  lDimSize = strtol(cp, &cp, 10);

  if (*cp != 0)
  {
    fprintf(stderr, "[ERROR] Argument '%s' is not an integer -- '%s'\n", argv[1], cp);
    exit(1);
  }

  *dimSize = (int)lDimSize;

  cp = argv[2];

  if (*cp == 0)
  {
    fprintf(stderr, "[ERROR] Argument %s is an empty string\n", argv[2]);
    exit(1);
  }

  lBlockSize = strtol(cp, &cp, 10);

  if (*cp != 0)
  {
    fprintf(stderr, "[ERROR] Argument '%s' is not an integer -- '%s'\n", argv[2], cp);
    exit(1);
  }

  *blockSize = (int)lBlockSize;
}

__global__ void getIndicesOnDevice(int *block, int *thread, int *index)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  block[idx] = blockIdx.x;
  thread[idx] = threadIdx.x;
  index[idx] = idx;
}

int main(int argc, char *argv[])
{
  int dimSize, blockSize, *h_block, *d_block, *h_thread, *d_thread, *h_index, *d_index;

  parseArgs(argc, argv, &dimSize, &blockSize);

  int N = dimSize * blockSize;

  size_t memSize = sizeof(int) * N;

  hipHostMalloc((void **)&h_block, memSize, hipHostMallocDefault);
  hipHostMalloc((void **)&h_thread, memSize, hipHostMallocDefault);
  hipHostMalloc((void **)&h_index, memSize, hipHostMallocDefault);

  hipMalloc((void **)&d_block, memSize);
  hipMalloc((void **)&d_thread, memSize);
  hipMalloc((void **)&d_index, memSize);

  getIndicesOnDevice<<<dimSize, blockSize>>>(d_block, d_thread, d_index);
  CUDACHECK(hipPeekAtLastError());

  hipMemcpy(h_block, d_block, memSize, hipMemcpyDeviceToHost);
  hipMemcpy(h_thread, d_thread, memSize, hipMemcpyDeviceToHost);
  hipMemcpy(h_index, d_index, memSize, hipMemcpyDeviceToHost);

  printf("\nDim Size: %6d   Block Size: %6d\n", dimSize, blockSize);

  printf("-------------------------------------\n");
  printf("|      i |  block | thread |  index |\n");
  printf("-------------------------------------\n");

  for (int i = 0; i < N; i++)
  {
    printf("| %6d | %6d | %6d | %6d |\n", i, h_block[i], h_thread[i], h_index[i]);
  }

  printf("-------------------------------------\n");
  printf("                            N: %6d\n\n", N);

  hipHostFree(h_block);
  hipFree(d_block);

  return 0;
}
