
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define CUDACHECK(err)                    \
  do                                      \
  {                                       \
    cudaCheck((err), __FILE__, __LINE__); \
  } while (false)

inline void cudaCheck(hipError_t error_code, const char *file, int line)
{
  if (error_code != hipSuccess)
  {
    fprintf(stderr, "CUDA Error %d: '%s'. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
    fflush(stderr);
    exit(error_code);
  }
}

void parseArgsInt(char *arg, int *val)
{
  char *cp;
  long lVal;

  cp = arg;

  if (*cp == 0)
  {
    fprintf(stderr, "[ERROR] Argument '%s' is an empty string\n", arg);
    exit(1);
  }

  lVal = strtol(cp, &cp, 10);

  if (*cp != 0)
  {
    fprintf(stderr, "[ERROR] Argument '%s' is not an integer -- '%s'\n", arg, cp);
    exit(1);
  }

  *val = (int)lVal;
}

void parseArgs(int argc, char *argv[], int *dimSize, int *blockSize)
{
  // Check for the right number of arguments
  if (argc != 3)
  {
    fprintf(stderr, "[ERROR] Must be run with exactly 2 argument, found %d!\nUsage: %s <dimSize> <blockSize>\n", argc - 1, argv[0]);
    exit(1);
  }

  parseArgsInt(argv[1], dimSize);
  parseArgsInt(argv[2], blockSize);
}

__global__ void getIndicesOnDevice(int *block, int *thread, int *index)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  block[idx] = blockIdx.x;
  thread[idx] = threadIdx.x;
  index[idx] = idx;
}

int main(int argc, char *argv[])
{
  int dimSize, blockSize, *h_block, *d_block, *h_thread, *d_thread, *h_index, *d_index;

  parseArgs(argc, argv, &dimSize, &blockSize);

  int N = dimSize * blockSize;

  size_t memSize = sizeof(int) * N;

  hipHostMalloc((void **)&h_block, memSize, hipHostMallocDefault);
  hipHostMalloc((void **)&h_thread, memSize, hipHostMallocDefault);
  hipHostMalloc((void **)&h_index, memSize, hipHostMallocDefault);

  hipMalloc((void **)&d_block, memSize);
  hipMalloc((void **)&d_thread, memSize);
  hipMalloc((void **)&d_index, memSize);

  getIndicesOnDevice<<<dimSize, blockSize>>>(d_block, d_thread, d_index);
  CUDACHECK(hipPeekAtLastError());

  hipMemcpy(h_block, d_block, memSize, hipMemcpyDeviceToHost);
  hipMemcpy(h_thread, d_thread, memSize, hipMemcpyDeviceToHost);
  hipMemcpy(h_index, d_index, memSize, hipMemcpyDeviceToHost);

  printf("\nDim Size: %6d   Block Size: %6d\n", dimSize, blockSize);

  printf("-------------------------------------\n");
  printf("|      i |  block | thread |  index |\n");
  printf("-------------------------------------\n");

  for (int i = 0; i < N; i++)
  {
    printf("| %6d | %6d | %6d | %6d |\n", i, h_block[i], h_thread[i], h_index[i]);
  }

  printf("-------------------------------------\n");
  printf("                            N: %6d\n\n", N);

  hipHostFree(h_block);
  hipFree(d_block);

  return 0;
}