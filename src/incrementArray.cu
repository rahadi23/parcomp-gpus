#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define CUDACHECK(err)                    \
  do                                      \
  {                                       \
    cudaCheck((err), __FILE__, __LINE__); \
  } while (false)

inline void cudaCheck(hipError_t error_code, const char *file, int line)
{
  if (error_code != hipSuccess)
  {
    fprintf(stderr, "CUDA Error %d: '%s'. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
    fflush(stderr);
    exit(error_code);
  }
}

void parseArgsInt(char *arg, int *val)
{
  char *cp;
  long lVal;

  cp = arg;

  if (*cp == 0)
  {
    fprintf(stderr, "[ERROR] Argument '%s' is an empty string\n", arg);
    exit(1);
  }

  lVal = strtol(cp, &cp, 10);

  if (*cp != 0)
  {
    fprintf(stderr, "[ERROR] Argument '%s' is not an integer -- '%s'\n", arg, cp);
    exit(1);
  }

  *val = (int)lVal;
}

void parseArgs(int argc, char *argv[], int *NMin, int *NMax, int *NInc, int *blockMin, int *blockMax, int *blockInc)
{
  // Check for the right number of arguments
  if (argc != 7)
  {
    fprintf(stderr, "[ERROR] Must be run with exactly 6 argument, found %d!\nUsage: %s <NMin> <NMax> <NInc> <blockMin> <blockMax> <blockInc>\n", argc - 1, argv[0]);
    exit(1);
  }

  parseArgsInt(argv[1], NMin);
  parseArgsInt(argv[2], NMax);
  parseArgsInt(argv[3], NInc);
  parseArgsInt(argv[4], blockMin);
  parseArgsInt(argv[5], blockMax);
  parseArgsInt(argv[6], blockInc);
}

void incrementArrayOnHost(float *a, int N)
{
  int i;
  for (i = 0; i < N; i++)
  {
    a[i] = a[i] + 1.f;
  }
}

__global__ void incrementArrayOnDevice(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N)
  {
    a[idx] = a[idx] + 1.f;
  }
}

int main(int argc, char *argv[])
{
  int j, k, NMin, NMax, NInc, NIter, blockMin, blockMax, blockInc, blockIter;

  parseArgs(argc, argv, &NMin, &NMax, &NInc, &blockMin, &blockMax, &blockInc);

  NIter = (NMax - NMin) / NInc + 1;
  blockIter = (blockMax - blockMin) / blockInc + 1;

  printf("\n-------------------------------------------------------------------------------\n");
  printf("|         N |   dimSize | blockSize |      isOk |      gpuTime |      cpuTime |\n");
  printf("-------------------------------------------------------------------------------\n");

  for (k = 0; k < NIter; k++)
  {
    for (j = 0; j < blockIter; j++)
    {
      float *a_h, *b_h; // pointers to host memory
      int i, N = NMin + k * NInc, blockSize = blockMin + j * blockInc;
      size_t size = N * sizeof(float);

      // allocate arrays on host
      a_h = (float *)malloc(size);
      b_h = (float *)malloc(size);

      // initialization of host data
      for (i = 0; i < N; i++)
      {
        a_h[i] = (float)i;
      }

      float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

      // some events to count the execution time
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      // start to count execution
      hipEventRecord(start, 0);

      // allocate array on device
      float *a_d; // pointer to device memory
      hipMalloc((void **)&a_d, size);

      // copy data from host to device
      hipMemcpy(a_d, a_h, sizeof(float) * N, hipMemcpyHostToDevice);

      // do calculation on device:
      // Part 1 of 2. Compute execution configuration
      int dimSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

      // Part 2 of 2. Call incrementArrayOnDevice kernel
      incrementArrayOnDevice<<<dimSize, blockSize>>>(a_d, N);
      CUDACHECK(hipPeekAtLastError());

      // Retrieve result from device and store in b_h
      hipMemcpy(b_h, a_d, sizeof(float) * N, hipMemcpyDeviceToHost);

      hipDeviceSynchronize();
      // time counting terminate
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);

      // compute time elapse on GPU computing
      hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);

      // start the CPU version
      hipEventRecord(start, 0);

      // do calculation on host
      incrementArrayOnHost(a_h, N);

      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);

      // check results
      bool resultIsOk = true;

      for (i = 0; i < N; i++)
      {
        resultIsOk = a_h[i] == b_h[i];
      }

      // assert(resultIsOk);
      printf("| %9d | %9d | %9d | %9d | %12.8f | %12.8f |\n", N, dimSize, blockSize, resultIsOk, gpu_elapsed_time_ms, cpu_elapsed_time_ms);

      // cleanup
      free(a_h);
      free(b_h);
      hipFree(a_d);
    }
  }

  printf("-------------------------------------------------------------------------------\n\n");
}
