#include "hip/hip_runtime.h"
/*
 * Implementation of conjugate-gradient for symmetric PSD systems using CUDA.
 *
 * Author: Tim Lebailly
 */
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>
#include <time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <sys/stat.h>

#include "../utils/helper.cuh"

extern "C"
{
#include "utils/helper.h"
#include "utils/sequential.h"
#include "../utils/helper.h"
}

// vecVec
#define BLOCK_DIM_VEC 1024

// matVec
#define NB_ELEM_MAT 32
#define BLOCK_SIZE_MAT 32

#define LOG_FILE_FORMAT "logs/conjugateGradient-%d.csv"

/*
 * --Naive implementation--
 * Computes a (square) matrix vector product
 * Input: pointer to 1D-array-stored matrix, 1D-array-stored vector
 * Stores the product in memory at the location of the pointer out
 */
__global__ void matVec(float *A, float *b, float *out, int N)
{
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < N)
	{
		float tmp = 0;
		for (int i = 0; i < N; i++)
		{
			tmp += b[i] * A[N * index_x + i];
		}
		out[index_x] = tmp;
	}
}

/*
 * --More efficient implementation--
 * Computes a (square) symmetric matrix vector product
 * Input: pointer to 1D-array-stored matrix, 1D-array-stored vector
 * Stores the product in memory at the location of the pointer out
 */
__global__ void matVec2(float *A, float *b, float *out, int N)
{
	__shared__ float b_shared[NB_ELEM_MAT];

	int effective_block_width;
	if ((blockIdx.x + 1) * NB_ELEM_MAT <= N)
	{
		effective_block_width = NB_ELEM_MAT;
	}
	else
	{
		// needed to avoid overflow in next row
		effective_block_width = N % NB_ELEM_MAT;
	}

	if (threadIdx.x < effective_block_width)
		b_shared[threadIdx.x] = b[blockIdx.x * NB_ELEM_MAT + threadIdx.x];

	__syncthreads();

	int idy = blockIdx.y * BLOCK_SIZE_MAT + threadIdx.x;
	float tmp_scal = 0.0;
	// threads outside matrix dimension are not needed (vertical)
	if (idy < N)
	{
		for (int i = 0; i < effective_block_width; i++)
		{
			// take advantage of symmetric matrix for coalesced memory access
			tmp_scal += b_shared[i] * A(blockIdx.x * NB_ELEM_MAT + i, idy, N);
		}
		atomicAdd(out + idy, tmp_scal);
	}
}

/*
 * Computes the sum of 2 vectors
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 * Stores the sum in memory at the location of the pointer out
 */
__global__ void vecPlusVec(float *a, float *b, float *out, int N)
{
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < N)
	{
		out[index_x] = b[index_x] + a[index_x];
	}
}

/*
 * Computes the sum of 2 vectors
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 * Stores the sum in memory at the location of the pointer out
 * Also 0's the vector b
 */
__global__ void vecPlusVec2(float *a, float *b, float *out, int N)
{
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < N)
	{
		out[index_x] = b[index_x] + a[index_x];
		b[index_x] = 0.0;
	}
}

/*
 * Computes the difference of 2 vectors
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 * Stores the sum in memory at the location of the pointer out
 */
__global__ void vecMinVec(float *a, float *b, float *out, int N)
{
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < N)
	{
		out[index_x] = a[index_x] - b[index_x];
	}
}

/*
 * --Naive implementation--
 * Computes the inner product of 2 vectors
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 * Stores the product in memory at the location of the pointer out
 */
__global__ void vecVec(float *a, float *b, float *out, int N)
{
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	float tmp = 0.0;
	if (index_x == 0)
	{
		for (int i = 0; i < N; i++)
		{
			tmp += b[i] * a[i];
		}
		*out = tmp;
	}
}

/*
 * --More efficient implementation--
 * Computes the inner product of 2 vectors
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 * Stores the product in memory at the location of the pointer out
 */
__global__ void vecVec2(float *a, float *b, float *out, int N)
{
	// each block has it's own shared_tmp of size BLOCK_DIM_VEC
	__shared__ float shared_tmp[BLOCK_DIM_VEC];

	// needed for atomicAdd
	if (threadIdx.x + blockDim.x * blockIdx.x == 0)
	{
		*out = 0.0;
	}

	if (blockIdx.x * blockDim.x + threadIdx.x < N)
	{
		shared_tmp[threadIdx.x] = a[blockIdx.x * blockDim.x + threadIdx.x] * b[blockIdx.x * blockDim.x + threadIdx.x];
	}
	else
	{
		// needed for the reduction
		shared_tmp[threadIdx.x] = 0.0;
	}

	// reduction within block
	for (int i = blockDim.x / 2; i >= 1; i = i / 2)
	{
		// threads access memory position written by other threads so sync is needed
		__syncthreads();
		if (threadIdx.x < i)
		{
			shared_tmp[threadIdx.x] += shared_tmp[threadIdx.x + i];
		}
	}

	// atomic add the partial reduction in out
	if (threadIdx.x == 0)
	{
		atomicAdd(out, shared_tmp[0]);
	}
}

/*
 * Computes the product of a scalar with a vector
 * Input: pointer to scalar, pointer to 1D-array-stored vector
 * Stores the sum in memory at the location of the pointer out
 */
__global__ void scalarVec(float *scalar, float *a, float *out, int N)
{
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < N)
	{
		out[index_x] = a[index_x] * *scalar;
	}
}

/*
 * Copies the content of vector in to vector out
 * Input: pointer to 1D-array-stored vector, pointer to 1D-array-stored vector
 */
__global__ void memCopy(float *in, float *out, int N)
{
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x < N)
	{
		out[index_x] = in[index_x];
	}
}

/*
 * Computes the quotient of 2 scalars
 * Input: pointer to scalar, pointer to scalar
 * Stores the quotient in memory at the location of the pointer out
 */
__global__ void divide(float *num, float *den, float *out)
{
	unsigned int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	if (index_x == 0)
	{
		*out = *num / *den;
	}
}

/*
 * Main CG solver
 * All the given pointers are device pointers, with correct initial values
 */
void solveCG_cuda(float *A, float *b, float *x, float *p, float *r, float *temp,
									float *alpha, float *beta, float *r_norm, float *r_norm_old,
									float *temp_scal, float *h_x, float *h_r_norm, int *cnt,
									int N, int maxIter, float eps)
{

	dim3 vec_block_dim(BLOCK_DIM_VEC);
	dim3 vec_grid_dim((N + BLOCK_DIM_VEC - 1) / BLOCK_DIM_VEC);

	dim3 mat_grid_dim((N + NB_ELEM_MAT - 1) / NB_ELEM_MAT, (N + BLOCK_SIZE_MAT - 1) / BLOCK_SIZE_MAT);
	dim3 mat_block_dim(BLOCK_SIZE_MAT);

	vecVec2<<<vec_grid_dim, vec_block_dim>>>(r, r, r_norm_old, N);
	int k = 0;
	while ((k < maxIter) && (*h_r_norm > eps))
	{
		// temp = A * p (only compute matrix vector product once)
		matVec2<<<mat_grid_dim, mat_block_dim>>>(A, p, temp, N);

		// alpha_k = ...
		vecVec2<<<vec_grid_dim, vec_block_dim>>>(p, temp, temp_scal, N);
		divide<<<1, 1>>>(r_norm_old, temp_scal, alpha);

		// r_{k+1} = ...
		scalarVec<<<vec_grid_dim, vec_block_dim>>>(alpha, temp, temp, N);
		vecMinVec<<<vec_grid_dim, vec_block_dim>>>(r, temp, r, N);

		// x_{k+1} = ...
		scalarVec<<<vec_grid_dim, vec_block_dim>>>(alpha, p, temp, N);
		vecPlusVec<<<vec_grid_dim, vec_block_dim>>>(x, temp, x, N);

		// beta_k = ...
		vecVec2<<<vec_grid_dim, vec_block_dim>>>(r, r, r_norm, N);
		divide<<<1, 1>>>(r_norm, r_norm_old, beta);

		// p_{k+1} = ...
		scalarVec<<<vec_grid_dim, vec_block_dim>>>(beta, p, temp, N);
		vecPlusVec2<<<vec_grid_dim, vec_block_dim>>>(r, temp, p, N);

		// set r_norm_old to r_norm
		memCopy<<<1, 1>>>(r_norm, r_norm_old, N);

		// copy to r_norm to CPU (to evaluate stop condition)
		hipMemcpy(h_r_norm, r_norm, sizeof(float), hipMemcpyDeviceToHost);
		k++;
	}

	*cnt = k;
	// printf("Time spent gpu per iter [s]: %e\n", (float)((micro_end_gpu - micro_begin_gpu) / k) / 1e6);
}

void parseArgs(int argc, char *argv[], int *NMin, int *NMax, int *NMult, int *MAX_ITER, float *EPS, float *TOL)
{
	// Check for the right number of arguments
	if (argc != 7)
	{
		fprintf(stderr, "[ERROR] Must be run with exactly 6 argument, found %d!\nUsage: %s <NMin> <NMax> <NMult> <MAX_ITER> <EPS> <TOL>\n", argc - 1, argv[0]);
		exit(1);
	}

	parseArgsInt(argv[1], NMin);
	parseArgsInt(argv[2], NMax);
	parseArgsInt(argv[3], NMult);
	parseArgsInt(argv[4], MAX_ITER);
	parseArgsFloat(argv[5], EPS);
	parseArgsFloat(argv[6], TOL);
}

////////////////////////////////////////////////////////////////////////////////
// MAIN
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[])
{
	int j, NMin, NMax, NMult, NIter, MAX_ITER;
	float EPS, TOL;

	parseArgs(argc, argv, &NMin, &NMax, &NMult, &MAX_ITER, &EPS, &TOL);

	struct stat buffer;

	int logId = 1;
	char *logFileNameWithId = (char *)malloc(sizeof(char) * strlen(LOG_FILE_FORMAT));

	do
	{
		sprintf(logFileNameWithId, LOG_FILE_FORMAT, logId);
		logId++;
	} while (stat(logFileNameWithId, &buffer) == 0);

	FILE *log_file = fopen(logFileNameWithId, "w");
	fprintf(log_file, "j,N,grid_size,block_size,is_ok,gpu_time,cpu_time,gpu_r_norm,cpu_r_norm,gpu_iter,cpu_iter,speedup\n");
	fclose(log_file);

	printf("\n----------------------------------------------------------------------------------------------------------------------------------------------------------------------\n");
	printf("|          N |   gridSize |  blockSize |      isOk |         gpuTime |         cpuTime |        gpuRNorm |        cpuRNorm |   gpuIter |   cpuIter |         speedUp |\n");
	printf("|            |   (nBlock) |  (nThread) |           |            (ms) |            (ms) |                 |                 |           |           |                 |\n");
	printf("----------------------------------------------------------------------------------------------------------------------------------------------------------------------\n");

	NIter = log10(NMax / NMin) / log10(NMult) + 1;

	for (j = 0; j < NIter; j++)
	{
		int N = N = NMin * pow(NMult, j);

		// allocate host memory
		float *h_A = generateA(N);
		float *h_b = generateb(N);
		float *h_x = (float *)calloc(N, sizeof(float));
		float *h_r_norm = (float *)malloc(sizeof(float));
		*h_r_norm = 1.0;

		// times
		int gpu_cnt, cpu_cnt;
		float cpu_r_norm, gpu_elapsed_time_ms, cpu_elapsed_time_ms;

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		// allocate device memory
		float *d_A;
		float *d_b;
		float *d_x;
		float *d_p;
		float *d_r;
		float *d_temp;
		hipMalloc((void **)&d_A, N * N * sizeof(float));
		hipMalloc((void **)&d_b, N * sizeof(float));
		hipMalloc((void **)&d_x, N * sizeof(float));
		hipMalloc((void **)&d_p, N * sizeof(float));
		hipMalloc((void **)&d_r, N * sizeof(float));
		hipMalloc((void **)&d_temp, N * sizeof(float));

		// copy host memory to device
		hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
		// assume x0 = 0
		hipMemcpy(d_p, h_b, N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_r, h_b, N * sizeof(float), hipMemcpyHostToDevice);

		// 4 floats needed
		float *d_beta;
		float *d_alpha;
		float *d_r_norm;
		float *d_r_norm_old;
		float *d_temp_scal;
		hipMalloc((void **)&d_beta, sizeof(float));
		hipMalloc((void **)&d_alpha, sizeof(float));
		hipMalloc((void **)&d_r_norm, sizeof(float));
		hipMalloc((void **)&d_r_norm_old, sizeof(float));
		hipMalloc((void **)&d_temp_scal, sizeof(float));

		hipEventRecord(start, 0);

		// run the main function
		solveCG_cuda(d_A, d_b, d_x, d_p, d_r, d_temp, d_alpha, d_beta, d_r_norm,
								 d_r_norm_old, d_temp_scal, h_x, h_r_norm, &gpu_cnt, N, MAX_ITER, EPS);

		CUDACHECK(hipPeekAtLastError());

		// allocate memory for the result on host side
		hipDeviceSynchronize();

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		// compute time elapse on GPU computing
		hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);

		// copy result from device to host
		hipMemcpy(h_x, d_x, sizeof(float) * N, hipMemcpyDeviceToHost);

		// compare output with sequential version
		float *h_x_seq = (float *)calloc(N, sizeof(float));

		hipEventRecord(start, 0);

		solveCG_seq(h_A, h_b, h_x_seq, &cpu_r_norm, &cpu_cnt, N, MAX_ITER, EPS);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		// compute time elapse on CPU computing
		hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);

		// assert(moreOrLessEqual(h_x, h_x_seq, N, TOL) == 1);
		int resultIsOk = moreOrLessEqual(h_x, h_x_seq, N, TOL) == 1;
		int gridSize = (N + BLOCK_DIM_VEC - 1) / BLOCK_DIM_VEC;
		int blockSize = BLOCK_DIM_VEC;
		float speedup = cpu_elapsed_time_ms / gpu_elapsed_time_ms;

		FILE *log_file = fopen(logFileNameWithId, "a");
		fprintf(log_file, "%d,%d,%d,%d,%d,%.6f,%.6f,%.6f,%.6f,%d,%d,%.6f\n",
						j, N, gridSize, blockSize, resultIsOk, gpu_elapsed_time_ms,
						cpu_elapsed_time_ms, *h_r_norm, cpu_r_norm, gpu_cnt, cpu_cnt, speedup);
		fclose(log_file);

		printf("| %10d | %10d | %10d | %9d | %15.6f | %15.6f | %15.9e | %15.9e | %9d | %9d | %15.6f |\n",
					 N, gridSize, blockSize, resultIsOk, gpu_elapsed_time_ms,
					 cpu_elapsed_time_ms, *h_r_norm, cpu_r_norm, gpu_cnt, cpu_cnt, speedup);

		// printf("\nAssertion passed!\n");

		// cleanup memory host
		free(h_A);
		free(h_b);
		free(h_x);
		free(h_r_norm);

		// cleanup memory device
		hipFree(d_A);
		hipFree(d_b);
		hipFree(d_x);
		hipFree(d_p);
		hipFree(d_r);
		hipFree(d_temp);
		hipFree(d_alpha);
		hipFree(d_beta);
		hipFree(d_r_norm);
		hipFree(d_r_norm_old);
		hipFree(d_temp_scal);
	}

	printf("----------------------------------------------------------------------------------------------------------------------------------------------------------------------\n\n");

	free(logFileNameWithId);

	return 0;
}
