#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/stat.h>

#include "../utils/helper.cuh"

extern "C"
{
#include "../utils/helper.h"
}

#define LOG_FILE_NAME "logs/matrixMultiplication.csv"

/*
*********************************************************************
function name: gpu_mult

description: dot product of two square matrix

parameters:
            &a GPU device pointer to a n X n matrix (A)
            &b GPU device pointer to a n X n matrix (B)
            &c GPU device output purpose pointer to a n X n matrix (C)
            to store the result

Note:
    grid and block should be configured as:
        dim3 dimGrid((k + blockSize - 1) / blockSize, (m + blockSize - 1) / blockSize);
        dim3 dimBlock(blockSize, blockSize);

return: none
*********************************************************************
*/
__global__ void gpu_mult(int *a, int *b, int *c, int N)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int sum = 0;
  if (col < N && row < N)
  {
    for (int i = 0; i < N; i++)
    {
      sum += a[row * N + i] * b[i * N + col];
    }
    c[row * N + col] = sum;
  }
}

/*
*********************************************************************
function name: gpu_mult_shared

description: dot product of two square matrix in GPU
             by using shared memory

parameters:
            &a GPU device pointer to a n X n matrix (A)
            &b GPU device pointer to a n X n matrix (B)
            &c GPU device output purpose pointer to a n X n matrix (C)
            to store the result
Note:
    grid and block should be configured as:

        dim3 dim_grid((n - 1) / blockSize + 1, (n - 1) / blockSize + 1, 1);
        dim3 dim_block(blockSize, blockSize, 1);

return: none
*********************************************************************
*/
__global__ void gpu_mult_shared(int *d_a, int *d_b, int *d_result, int N, int blockSize)
{
  extern __shared__ int tiles[];

  int *tile_a = tiles;
  int *tile_b = (int *)&tiles[blockSize * blockSize];

  int row = blockIdx.y * blockSize + threadIdx.y;
  int col = blockIdx.x * blockSize + threadIdx.x;
  int tmp = 0;
  int idx;

  for (int sub = 0; sub < gridDim.x; ++sub)
  {
    idx = row * N + sub * blockSize + threadIdx.x;
    if (idx >= N * N)
    {
      // n may not divisible by blockSize
      tile_a[threadIdx.y * blockSize + threadIdx.x] = 0;
    }
    else
    {
      tile_a[threadIdx.y * blockSize + threadIdx.x] = d_a[idx];
    }

    idx = (sub * blockSize + threadIdx.y) * N + col;
    if (idx >= N * N)
    {
      tile_b[threadIdx.y * blockSize + threadIdx.x] = 0;
    }
    else
    {
      tile_b[threadIdx.y * blockSize + threadIdx.x] = d_b[idx];
    }

    __syncthreads();

    for (int k = 0; k < blockSize; ++k)
    {
      tmp += tile_a[threadIdx.y * blockSize + k] * tile_b[k * blockSize + threadIdx.x];
    }

    __syncthreads();
  }

  if (row < N && col < N)
  {
    d_result[row * N + col] = tmp;
  }
}

/*
*********************************************************************
function name: cpu_mult

description: dot product of two square matrix in CPU,
             for validating GPU results

parameters:
            &a CPU host pointer to a N X N matrix (A)
            &b CPU host pointer to a N X N matrix (B)
            &c CPU host output purpose pointer to a N X N matrix (C)
            to store the result
return: none
*********************************************************************
*/
void cpu_mult(int *h_a, int *h_b, int *h_result, int N)
{
  for (int i = 0; i < N; ++i)
  {
    for (int j = 0; j < N; ++j)
    {
      int tmp = 0.0;
      for (int h = 0; h < N; ++h)
      {
        tmp += h_a[i * N + h] * h_b[h * N + j];
      }
      h_result[i * N + j] = tmp;
    }
  }
}

void parseArgs(int argc, char *argv[],
               int *NMin, int *NMax, int *NMult,
               int *blockMin, int *blockMax, int *blockMult)
{
  // Check for the right number of arguments
  if (argc != 7)
  {
    fprintf(stderr, "[ERROR] Must be run with exactly 6 argument, found %d!\nUsage: %s <NMin> <NMax> <NMult> <blockMin> <blockMax> <blockMult>\n", argc - 1, argv[0]);
    exit(1);
  }

  parseArgsInt(argv[1], NMin);
  parseArgsInt(argv[2], NMax);
  parseArgsInt(argv[3], NMult);
  parseArgsInt(argv[4], blockMin);
  parseArgsInt(argv[5], blockMax);
  parseArgsInt(argv[6], blockMult);
}

int main(int argc, char *argv[])
{
  int NMin, NMax;
  int NMult, NIter, blockMin, blockMax, blockMult, blockIter;

  parseArgs(argc, argv, &NMin, &NMax, &NMult, &blockMin, &blockMax, &blockMult);

  NIter = log10(NMax / NMin) / log10(NMult) + 1;
  blockIter = log10(blockMax / blockMin) / log10(blockMult) + 1;

  struct stat buffer;

  if (stat(LOG_FILE_NAME, &buffer) != 0)
  {
    FILE *log_file = fopen(LOG_FILE_NAME, "w");
    fprintf(log_file, "k,l,N,grid_size,block_size,is_ok,gpu_time,gpu_shared_time,cpu_time,gpu_speedup,gpu_shared_speedup\n");
    fclose(log_file);
  }

  printf("\n+-----------+-----------+-----------+-----------+--------------+--------------+--------------+--------------+--------------+\n");
  printf("|         N |  gridSize | blockSize |      isOk |      gpuTime |    gpuShTime |      cpuTime |   gpuSpeedUp | gpuShSpeedUp |\n");
  printf("|           |  (nBlock) | (nThread) |           |         (ms) |         (ms) |         (ms) |              |              |\n");
  printf("+-----------+-----------+-----------+-----------+--------------+--------------+--------------+--------------+--------------+\n");

  for (int iN = 0; iN < NIter; iN++)
  {
    float n_cpu_elapsed_time_ms = -1;

    for (int iBlock = 0; iBlock < blockIter; iBlock++)
    {
      int N = NMin * pow(NMult, iN);

      int blockSize = blockMin * pow(blockMult, iBlock);
      int gridSize = (N + blockSize - 1) / blockSize;

      dim3 dimGrid(gridSize, gridSize);
      dim3 dimBlock(blockSize, blockSize);

      printf("| %9d | %9d | %9d | ",
             N, gridSize, blockSize);

      size_t NNSize = sizeof(int) * N * N;

      // allocate memory in host RAM
      int *h_a, *h_b, *h_uc, *h_sc, *h_cc;
      hipHostMalloc((void **)&h_a, NNSize);
      hipHostMalloc((void **)&h_b, NNSize);
      hipHostMalloc((void **)&h_uc, NNSize);
      hipHostMalloc((void **)&h_sc, NNSize);
      hipHostMalloc((void **)&h_cc, NNSize);

      // random initialize matrix A
      for (int iRow = 0; iRow < N; ++iRow)
      {
        for (int iCol = 0; iCol < N; ++iCol)
        {
          h_a[iRow * N + iCol] = rand() % 1024;
        }
      }

      // random initialize matrix B
      for (int iRow = 0; iRow < N; ++iRow)
      {
        for (int iCol = 0; iCol < N; ++iCol)
        {
          h_b[iRow * N + iCol] = rand() % 1024;
        }
      }

      float gpu_elapsed_time_ms, gpu_shared_elapsed_time_ms, cpu_elapsed_time_ms;

      // some events to count the execution time
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      // start to count execution time of GPU version
      hipEventRecord(start, 0);

      // initialize GPU
      int *d_ua, *d_ub, *d_uc;

      // Allocate memory space on the device
      hipMalloc((void **)&d_ua, NNSize);
      hipMalloc((void **)&d_ub, NNSize);
      hipMalloc((void **)&d_uc, NNSize);

      // copy matrix A and B from host to device memory
      hipMemcpy(d_ua, h_a, NNSize, hipMemcpyHostToDevice);
      hipMemcpy(d_ub, h_b, NNSize, hipMemcpyHostToDevice);

      gpu_mult<<<dimGrid, dimBlock>>>(d_ua, d_ub, d_uc, N);
      CUDACHECK(hipPeekAtLastError());

      // Transfer results from device to host
      hipMemcpy(h_uc, d_uc, NNSize, hipMemcpyDeviceToHost);
      hipDeviceSynchronize();

      // time counting terminate
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);

      // compute time elapse on GPU computing
      hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);

      // start to count execution time of GPU with shared memory version
      hipEventRecord(start, 0);

      // initialize GPU
      int *d_sa, *d_sb, *d_sc;

      // Allocate memory space on the device
      hipMalloc((void **)&d_sa, NNSize);
      hipMalloc((void **)&d_sb, NNSize);
      hipMalloc((void **)&d_sc, NNSize);

      // copy matrix A and B from host to device memory
      hipMemcpy(d_sa, h_a, NNSize, hipMemcpyHostToDevice);
      hipMemcpy(d_sb, h_b, NNSize, hipMemcpyHostToDevice);

      gpu_mult_shared<<<dimGrid,
                        dimBlock,
                        blockSize * blockSize * sizeof(int) * 2>>>(
          d_sa, d_sb, d_sc, N, blockSize);
      CUDACHECK(hipPeekAtLastError());

      // Transfer results from device to host
      hipMemcpy(h_sc, d_sc, NNSize, hipMemcpyDeviceToHost);
      hipDeviceSynchronize();

      // time counting terminate
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);

      // compute time elapse on GPU with shared memory computing
      hipEventElapsedTime(&gpu_shared_elapsed_time_ms, start, stop);

      // start the CPU version
      hipEventRecord(start, 0);

      cpu_mult(h_a, h_b, h_cc, N);

      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);

      if (n_cpu_elapsed_time_ms < 0 || cpu_elapsed_time_ms < n_cpu_elapsed_time_ms)
      {
        n_cpu_elapsed_time_ms = cpu_elapsed_time_ms;
      }

      // validate results computed by GPU
      int resultIsOk = 1;

      for (int i = 0; i < N; ++i)
      {
        for (int j = 0; j < N; ++j)
        {
          // printf("[%d][%d]:%d == [%d][%d]:%d, [%d][%d]:%d == [%d][%d]:%d\n",
          //        i, j, h_cc[i * k + j], i, j, h_uc[i * k + j],
          //        i, j, h_cc[i * k + j], i, j, h_sc[i * k + j]);

          if (h_cc[i * N + j] != h_uc[i * N + j] || h_cc[i * N + j] != h_sc[i * N + j])
          {
            resultIsOk = 0;
          }
        }
      }

      float gpu_speedup = n_cpu_elapsed_time_ms / gpu_elapsed_time_ms,
            gpu_shared_speedup = n_cpu_elapsed_time_ms / gpu_shared_elapsed_time_ms;

      FILE *log_file = fopen(LOG_FILE_NAME, "a");
      fprintf(log_file, "%d,%d,%d,%d,%d,%d,%.6f,%.6f,%.6f,%.6f,%.6f\n",
              iN, iBlock, N, gridSize, blockSize, resultIsOk,
              gpu_elapsed_time_ms, gpu_shared_elapsed_time_ms,
              n_cpu_elapsed_time_ms, gpu_speedup, gpu_shared_speedup);
      fclose(log_file);

      printf("%9d | %12.6f | %12.6f | %12.6f | %12.6f | %12.6f |\n",
             resultIsOk,
             gpu_elapsed_time_ms, gpu_shared_elapsed_time_ms,
             n_cpu_elapsed_time_ms, gpu_speedup, gpu_shared_speedup);

      // free memory
      hipFree(d_ua);
      hipFree(d_ub);
      hipFree(d_uc);
      hipFree(d_sa);
      hipFree(d_sb);
      hipFree(d_sc);
      hipHostFree(h_a);
      hipHostFree(h_b);
      hipHostFree(h_uc);
      hipHostFree(h_sc);
      hipHostFree(h_cc);
    }
  }

  printf("+-----------+-----------+-----------+-----------+--------------+--------------+--------------+--------------+--------------+\n\n");

  return 0;
}
